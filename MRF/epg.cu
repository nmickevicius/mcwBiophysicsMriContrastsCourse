
#include <hip/hip_runtime.h>

#define STATES 2056

__device__ void transitionMatrix(float Tr[3][3], float Ti[3][3], float alpha, float phi)
{
    float halpha = 0.5*alpha;

    // real part 
    Tr[0][0] = cos(halpha)*cos(halpha);
    Tr[0][1] = sin(halpha)*sin(halpha)*cos(2.0*phi);// + 1j*np.sin(halpha)*np.sin(halpha)*np.sin(2*phi)
    Tr[0][2] = sin(alpha)*sin(phi);// - 1j*np.sin(alpha)*np.cos(phi)
    Tr[1][0] = sin(halpha)*sin(halpha)*cos(2.0*phi); //- 1j*np.sin(halpha)*np.sin(halpha)*np.sin(2.0*phi)
    Tr[1][1] = cos(halpha)*cos(halpha);
    Tr[1][2] = sin(alpha)*sin(phi);// + 1j*np.sin(alpha)*np.cos(phi)
    Tr[2][0] = -0.5*sin(alpha)*sin(phi);// - 1j*0.5*np.sin(alpha)*np.cos(phi)
    Tr[2][1] = -0.5*sin(alpha)*sin(phi);// + 1j*0.5*np.sin(alpha)*np.cos(phi)
    Tr[2][2] = cos(alpha);

    // imaginary part 
    Ti[0][0] = 0.0f;
    Ti[0][1] = sin(halpha)*sin(halpha)*sin(2*phi);
    Ti[0][2] = -sin(alpha)*cos(phi);
    Ti[1][0] = -sin(halpha)*sin(halpha)*sin(2.0*phi);
    Ti[1][1] = 0.0f;
    Ti[1][2] = sin(alpha)*cos(phi);
    Ti[2][0] = -0.5*sin(alpha)*cos(phi);
    Ti[2][1] = 0.5*sin(alpha)*cos(phi);
    Ti[2][2] = 0.0f;
}

__device__ void rf(float omr[3][STATES], float omi[3][STATES], float alpha, float phi, float Tr[3][3], float Ti[3][3], int nmax) 
{

    float xr[3], xi[3];

    // calculate the transition matrix 
    transitionMatrix(Tr, Ti, alpha, phi);

    // only apply to maximum number of configuration states 
    for (int i=0; i<nmax; i++) {

        for (int j=0; j<3; j++) { 
            xr[j] = 0.0f;
            xi[j] = 0.0f;
            for (int k=0; k<3; k++) { 
                xr[j] += (Tr[j][k]*omr[k][i] - Ti[j][k]*omi[k][i]);
                xi[j] += (Tr[j][k]*omi[k][i] + Ti[j][k]*omr[k][i]);
            }
        }
        for (int j=0; j<3; j++) {
            omr[j][i] = xr[j];
            omi[j][i] = xi[j];
        }
    }

}

__device__ void relax(float omr[3][STATES], float omi[3][STATES], float t1, float t2, float dt, int nmax)
{
    float E1 = exp(-dt/t1);
    float E2 = exp(-dt/t2);
    for (int n=0; n<nmax; n++) {
        omr[0][n] *= E2;
        omi[0][n] *= E2;
        omr[1][n] *= E2;
        omi[1][n] *= E2;
        omr[2][n] *= E1;
        omi[2][n] *= E1;
    }
    omr[2][0] += (1 - E1);
}   

__device__ void spoil(float omr[3][STATES], float omi[3][STATES], float omrcpy[3][STATES], float omicpy[3][STATES], int nmax)
{
    int i, j;
    for (i=0; i<2; i++) {
        for (j=0; j<nmax; j++) {
            omrcpy[i][j] = omr[i][j];
            omicpy[i][j] = omi[i][j];
        }
    }
    for (i=0; i<nmax; i++) {
        omr[0][i+1] = omrcpy[0][i];
        omi[0][i+1] = omicpy[0][i];
        omr[1][i] = omrcpy[1][i+1];
        omi[1][i] = omicpy[1][i+1];
    }
    omr[0][0] = omr[1][0];
    omi[0][0] = -omi[1][0];
}

__device__ void sample(float *yr, float *yi, float omr[3][STATES], float omi[3][STATES], float phi)
{
    *yr = omr[0][0]*cos(phi) - omi[0][0]*sin(phi);
    *yi = omr[0][0]*sin(phi) + omi[0][0]*cos(phi);
}



__global__ void gre_epg_cuda(float *destr, float *desti, float *alpha, float *phi, float *tr, float *t1, float *t2, int nrf, int nt1, int isIR)
{

    int n = blockIdx.x*blockDim.x + threadIdx.x;

    if ((n >= 0) && (n < nt1)) {

        // initialize magnetization 
        float omr[3][STATES];
        float omi[3][STATES];
        for (int i=0; i<3; i++) {
            for (int j=0; j<STATES; j++) {
                omr[i][j] = 0.0f;
                omi[i][j] = 0.0f;
            }
        }
        omr[2][0] = 1.0f;
        if (isIR) 
            omr[2][0] = -1.0f;

        // allocate another configuration matrix for fast spoiling operations 
        float omr_copy[3][STATES];
        float omi_copy[3][STATES];

        // real and imaginary part of RF pulse transition matrix 
        float Tr[3][3], Ti[3][3];

        // loop over RF pulses 
        for (int p=0; p<nrf; p++) {

            // apply the RF pulse 
            rf(omr, omi, alpha[p], phi[p], Tr, Ti, p+1);

            // sample the signal and store in output
            sample(&destr[n*nrf + p], &desti[n*nrf + p], omr, omi, -phi[p]);
            
            // relaxation until TR 
            relax(omr, omi, t1[n], t2[n], tr[p], p+1);

            // gradient spoiling 
            spoil(omr, omi, omr_copy, omi_copy, p+2);

        } // end pulse loop 

    }

}